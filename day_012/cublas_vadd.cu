
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>

int main() {
    const int N = 1024;
    float A[N], B[N], C[N];

    // initialization
    for (int i = 0; i < N; i++) {
        A[i] = i;
        B[i] = i;
    }

    // create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // allocate device memory
    float *d_a, *d_b;
    hipMalloc((void**)&d_a, N*sizeof(float));
    hipMalloc((void**)&d_b, N*sizeof(float));

    // copy data from host to device
    hipMemcpy(d_a, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, N * sizeof(float), hipMemcpyHostToDevice);

    // scaling factor
    const float alpha =2.0f;

    // perform vector addition
    hipblasSaxpy(handle, N, &alpha, d_a, 1, d_b, 1);

    // copy result back to host (result is in d_b)
    hipMemcpy(C, d_b, N * sizeof(float), hipMemcpyDeviceToHost);

    // print results
    for (int i = 0; i < N; i++) {
        printf("%f ", C[i]);
    }

    // cleanup
    hipFree(d_a); hipFree(d_b);
    hipblasDestroy(handle);

    return 0;
}

